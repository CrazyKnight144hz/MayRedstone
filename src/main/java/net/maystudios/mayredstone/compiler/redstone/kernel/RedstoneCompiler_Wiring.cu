
#include <hip/hip_runtime.h>
extern "C"

__device__ void checkNeighbour(int x, int y, int z, int *blockType, int *blockState, int *blockAdditional, int *blockAdvanced, int *blockDirection,
                               int *newBlockType, int *newBlockState, int *newBlockAdditional, int*newBlockAdvanced, int *newBlockDirection) {

    int zLength = (gridDim.z * blockDim.z);
    int xzLength = (gridDim.y * blockDim.y) * zLength;

    // X
    if(x - 1 > 0) {
        if (blockType[xzLength * (x - 1) + zLength * y + z] > 1 && blockState[xzLength * (x - 1) + zLength * y + z] == 1) {
            if (blockType[xzLength * (x - 1) + zLength * y + z] == 200) {
                newBlockAdditional[xzLength * (x - 1) + zLength * y + z] = newBlockAdditional[xzLength * (x - 1) + zLength * y + z] - 1;
            } else {
                newBlockAdditional[xzLength * (x - 1) + zLength * y + z] = 15;
            }
        }
    }

    if(x + 1 < gridDim.x * blockDim.x) {
        if (blockType[xzLength * (x + 1) + zLength * y + z] > 1 && blockState[xzLength * (x + 1) + zLength * y + z] == 1) {
            if (blockType[xzLength * (x + 1) + zLength * y + z] == 200) {
                newBlockAdditional[xzLength * (x + 1) + zLength * y + z] = newBlockAdditional[xzLength * (x + 1) + zLength * y + z] - 1;
            } else {
                newBlockAdditional[xzLength * (x + 1) + zLength * y + z] = 15;
            }
        }
    }


    // Y
    if(y - 1 > 0) {
        if (blockType[xzLength * x + zLength * (y - 1) + z] > 1 && blockState[xzLength * x + zLength * (y - 1) + z] == 1) {
            if (blockType[xzLength * x + zLength * (y - 1) + z] == 200) {
                newBlockAdditional[xzLength * x + zLength * (y - 1) + z] = newBlockAdditional[xzLength * x + zLength * (y - 1) + z] - 1;
            } else {
                newBlockAdditional[xzLength * x + zLength * (y - 1) + z] = 15;
            }
        }
    }

    if(y + 1 < gridDim.y * blockDim.y) {
        if (blockType[xzLength * x + zLength * (y + 1) + z] > 1 && blockState[xzLength * x + zLength * (y + 1) + z] == 1) {
            if (blockType[xzLength * x + zLength * (y + 1) + z] == 200) {
                newBlockAdditional[xzLength * x + zLength * (y + 1) + z] = newBlockAdditional[xzLength * x + zLength * (y + 1) + z] - 1;
            } else {
                newBlockAdditional[xzLength * x + zLength * (y + 1) + z] = 15;
            }
        }
    }


    // Z
    if(z - 1 > 0) {
        if (blockType[xzLength * x + zLength * y + (z - 1)] > 1 && blockState[xzLength * x + zLength * y + (z - 1)] == 1) {
            if (blockType[xzLength * x + zLength * y + (z - 1)] == 200) {
                newBlockAdditional[xzLength * x + zLength * y + (z - 1)] = newBlockAdditional[xzLength * x + zLength * y + (z - 1)] - 1;
            } else {
                newBlockAdditional[xzLength * x + zLength * y + (z - 1)] = 15;
            }
        }
    }

    if(z + 1 < gridDim.z * blockDim.z) {
        if (blockType[xzLength * x + zLength * y + (z + 1)] > 1 && blockState[xzLength * x + zLength * y + (z + 1)] == 1) {
            if (blockType[xzLength * x + zLength * y + (z + 1)] == 200) {
                newBlockAdditional[xzLength * x + zLength * y + (z + 1)] = newBlockAdditional[xzLength * x + zLength * y + (z + 1)] - 1;
            } else {
                newBlockAdditional[xzLength * x + zLength * y + (z + 1)] = 15;
            }
        }
    }
}

__global__ void RedstoneCompiler_Wiring(int *blockType, int *blockState, int *blockAdditional, int *blockAdvanced, int *blockDirection,
                                        int *newBlockType, int *newBlockState, int *newBlockAdditional, int*newBlockAdvanced, int *newBlockDirection) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int zLength = (gridDim.z * blockDim.z);
    int xzLength = (gridDim.y * blockDim.y) * zLength;

    if(blockType[xzLength * x + zLength * y + z] != 200) {
        return;
    }

    checkNeighbour(x, y, z, blockType, blockState, blockAdditional, blockAdvanced, blockDirection, newBlockType, newBlockState, newBlockAdditional, newBlockAdvanced, newBlockDirection);
}

