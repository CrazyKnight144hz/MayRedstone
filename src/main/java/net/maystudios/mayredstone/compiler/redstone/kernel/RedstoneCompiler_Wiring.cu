#include "hip/hip_runtime.h"
extern "C"

struct Block {
    int BlockType;
    int BlockState;
    int BlockAdditional;
    int BlockAdvanced;
    int BlockDirection;
};

__global__ void RedstoneCompiler_Wiring(Block *blocks, Blocks* newBlocks) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int zLength = (gridDim.z * blockDim.z);
    int xzLength = (gridDim.y * blockDim.y) * zLength;

    int index = xzLength * x + zLength * y + z;

    if(blocks->BlockType != 200) {
        return;
    }
}

__device__ void checkNeighbour(Block *blocks, Blocks* newBlocks, int x, int y, int z) {

    int zLength = (gridDim.z * blockDim.z);
    int xzLength = (gridDim.y * blockDim.y) * zLength;

    // X
    if(x - 1 > 0) {
        if (blocks[xzLength * (x - 1) + zLength * y + z])->BlockType > 1 && blocks[xzLength * (x - 1) + zLength * y + z]->BlockState == 1) {
            if (blocks[] == 200) {
                newBlocks[xzLength * (x - 1) + zLength * y + z]->BlockAdditional = blocks[xzLength * (x - 1) + zLength * y + z]->BlockAdditional - 1;
            } else {
                newBlocks[xzLength * (x - 1) + zLength * y + z]->BlockAdditional = 15;
            }
        }
    }

    if(x + 1 < gridDim.x * blockDim.x) {
        if (blocks[xzLength * (x + 1) + zLength * y + z])->BlockType > 1 && blocks[xzLength * (x + 1) + zLength * y + z]->BlockState == 1) {
            if (blocks[] == 200) {
                newBlocks[xzLength * (x + 1) + zLength * y + z]->BlockAdditional = blocks[xzLength * (x + 1) + zLength * y + z]->BlockAdditional - 1;
            } else {
                newBlocks[xzLength * (x + 1) + zLength * y + z]->BlockAdditional = 15;
            }
        }
    }


    // Y
    if(y - 1 > 0) {
        if (blocks[xzLength * x + zLength * (y - 1) + z])->BlockType > 1 && blocks[xzLength * x + zLength * (y - 1) + z]->BlockState == 1) {
            if (blocks[] == 200) {
                newBlocks[xzLength * x + zLength * (y - 1) + z]->BlockAdditional = blocks[xzLength * x + zLength * (y - 1) + z]->BlockAdditional - 1;
            } else {
                newBlocks[xzLength * x + zLength * (y - 1) + z]->BlockAdditional = 15;
            }
        }
    }

    if(y + 1 < gridDim.y * blockDim.y) {
        if (blocks[xzLength * x + zLength * (y + 1) + z])->BlockType > 1 && blocks[xzLength * x + zLength * (y + 1) + z]->BlockState == 1) {
            if (blocks[] == 200) {
                newBlocks[xzLength * x + zLength * (y + 1) + z]->BlockAdditional = blocks[xzLength * x + zLength * (y + 1) + z]->BlockAdditional - 1;
            } else {
                newBlocks[xzLength * x + zLength * (y + 1) + z]->BlockAdditional = 15;
            }
        }
    }


    // Z
    if(z - 1 > 0) {
        if (blocks[xzLength * x + zLength * y + (z - 1)])->BlockType > 1 && blocks[xzLength * x + zLength * y + (z - 1)]->BlockState == 1) {
            if (blocks[] == 200) {
                newBlocks[xzLength * x + zLength * y + (z - 1)]->BlockAdditional = blocks[xzLength * x + zLength * y + (z - 1)]->BlockAdditional - 1;
            } else {
                newBlocks[xzLength * x + zLength * y + (z - 1)]->BlockAdditional = 15;
            }
        }
    }

    if(z + 1 < gridDim.z * blockDim.z) {
        if (blocks[xzLength * x + zLength * y + (z + 1)])->BlockType > 1 && blocks[xzLength * x + zLength * y + (z + 1)]->BlockState == 1) {
            if (blocks[] == 200) {
                newBlocks[xzLength * x + zLength * y + (z + 1)]->BlockAdditional = blocks[xzLength * x + zLength * y + (z + 1)]->BlockAdditional - 1;
            } else {
                newBlocks[xzLength * x + zLength * y + (z + 1)]->BlockAdditional = 15;
            }
        }
}