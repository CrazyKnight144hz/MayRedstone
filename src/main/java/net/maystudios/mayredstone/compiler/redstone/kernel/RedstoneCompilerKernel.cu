#include "hip/hip_runtime.h"
extern "C"
__global__ void RedstoneCompiler(int *Blocks, int *BlockStates, int *newBlockStates) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;


}

__device__ void (int x, int y , int z, int *Blocks, int *BlockStates, int *newBlockStates) {

}

__global__ void RedstoneDustCompiler(int *Blocks, int *BlockStates, int *newBlockStates, int x, int y, int z) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = x * (gridDim.y * blockDim.y ) + y
}