#include "hip/hip_runtime.h"
extern "C"

struct Block {
    int BlockType;
    int BlockState;
};

__global__ void RedstoneCompiler(Block *Blocks, Block *newBlockStates) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int zLength = (gridDim.z * blockDim.z);
    int xzLength = (gridDim.y * blockDim.y) * zLength;

    int index = xzLength * x + zLength * y + z;

    int _BlockType = Block[index]->BlockType;

    switch(BlockType) {
        case 0:

        break;

        case 1:

        break;
    }
}

__device__ void calc (int x, int y , int z, int *Blocks, int *BlockStates, int *newBlockStates) {

}



/*
__global__ void RedstoneDustCompiler(int *Blocks, int *BlockStates, int *newBlockStates, int x, int y, int z) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    int index = 0;

    if (z - 1 >= 0) {
        
    }
}

__global__ void RedstonePoweredCompiler(hipDeviceptr_t devBlocks, int *newBlockStates) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // int index = (gridDim.y * blockDim.y) * (gridDim.z * blockDim.z) * x + (gridDim.z * blockDim.z) * y + z;

    int zLength = (gridDim.z * blockDim.z);
    int xzLength = (gridDim.y * blockDim.y) * zLength;

    int index = xzLength * x + zLength * y + z;

    Block particle = ((Block*)devBlocks)[index];

    switch ()
}
*/

