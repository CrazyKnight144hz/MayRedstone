
#include <hip/hip_runtime.h>
extern "C"
__global__ void sampleKernel(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 2)
    {
        c[i] = a[i] + b[i];
    }
}